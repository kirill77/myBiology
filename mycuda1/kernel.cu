#include "hip/hip_runtime.h"
﻿#include "neural/tensor.h"
#include "neural/network.h"

#define RUN_ON_CPU 1

template <ACTIVATION T_ACTIVATION1, ACTIVATION T_ACTIVATION2>
struct FullyConnectedLayerCuda
{
    FullyConnectedLayerCuda(Tensor<float>& input, Tensor<float>& output, Tensor<float>& weights, Tensor<float>& biases) :
        m_input(input), m_output(output), m_weights(weights), m_biases(biases)
    {
    }

    void forward(unsigned blockX, unsigned blockY, unsigned threadX, unsigned threadY)
    {
        unsigned inOutNi = blockX;
        unsigned inOutCi = blockY;
        unsigned outWi = threadX;
        unsigned outHi = threadY;

        unsigned iBias = outHi / (T_ACTIVATION1 == T_ACTIVATION2 ? 1 : 2) * m_output.w() + outWi;
        unsigned iWeight = m_input.h() * m_input.w() * iBias;
        float fBeforeActivation = m_biases[iBias];
        for (unsigned inHi = 0; inHi < m_input.h(); ++inHi)
        {
            for (unsigned inWi = 0; inWi < m_input.w(); ++inWi)
            {
                fBeforeActivation += m_input.access(inOutNi, inHi, inWi, inOutCi) * m_weights[iWeight++];
            }
        }
        float fAfterActivation = TFunction<T_ACTIVATION1>(fBeforeActivation);
        m_output.access(inOutNi, outHi, outWi, inOutCi) = fAfterActivation;
        if (T_ACTIVATION1 != T_ACTIVATION2)
        {
            float fAfterActivation2 = TFunction<T_ACTIVATION2>(fBeforeActivation);
            m_output.access(inOutNi, outHi + 1, outWi, inOutCi) = fAfterActivation2;
        }
    }

    Tensor<float> m_input, m_weights, m_biases, m_output;
};

template <ACTIVATION T_ACTIVATION1, ACTIVATION T_ACTIVATION2>
__global__ void fullyConnectedLayerForward(FullyConnectedLayerCuda<T_ACTIVATION1, T_ACTIVATION2> p)
{
    p.forward(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

template <ACTIVATION T_ACTIVATION1, ACTIVATION T_ACTIVATION2>
void FullyConnectedLayer<T_ACTIVATION1, T_ACTIVATION2>::forward(std::vector<TensorRef>& inputs)
{
    nvAssert(inputs.size() == 1 && m_outputs.size() == 1); // this layer has one input tensor and one output tensor
    Tensor<float>& input = *inputs[0];
    nvAssert(input.n() == m_inputDims[0] && input.h() == m_inputDims[1] && input.w() == m_inputDims[2] && input.c() == m_inputDims[3]);
    Tensor<float>& output = *m_outputs[0];
    nvAssert(output.n() == m_outputDims[0] && output.h() == m_outputDims[1] && output.w() == m_outputDims[2] && output.c() == m_outputDims[3]);

    dim3 grid(m_outputDims[0], m_outputDims[3], 1);
    dim3 block(m_outputDims[2], T_ACTIVATION1 == T_ACTIVATION2 ? m_outputDims[1] : m_outputDims[1] / 2, 1);
#if RUN_ON_CPU
    FullyConnectedLayerCuda<T_ACTIVATION1, T_ACTIVATION2> cudaLayer(input, output, m_weights, m_biases);
    for (unsigned iBlockY = 0; iBlockY < grid.y; ++iBlockY)
    {
        for (unsigned iBlockX = 0; iBlockX < grid.x; ++iBlockX)
        {
            for (unsigned iThreadY = 0; iThreadY < block.y; ++iThreadY)
            {
                for (unsigned iThreadX = 0; iThreadX < block.x; ++iThreadX)
                {
                    cudaLayer.forward(iBlockX, iBlockY, iThreadX, iThreadY);
                }
            }
        }
    }
#else
    
#endif
}

template struct FullyConnectedLayer<ACTIVATION_RELU, ACTIVATION_MRELU>;
template struct FullyConnectedLayer<ACTIVATION_IDENTITY, ACTIVATION_IDENTITY>;