#include "hip/hip_runtime.h"
#include "neural/tensor.h"
#include "neural/network.h"
#include "neural/atomsNetwork.h"

__host__ void _myCheckCudaErrors()
{
    hipError_t status = hipDeviceSynchronize();
    nvAssert(status == hipSuccess);
}

size_t g_nCudaBytes = 0;
__host__ hipError_t myCudaMalloc(void** devPtr, size_t size)
{
    g_nCudaBytes += size;
    auto result = hipMalloc(devPtr, size);
    nvAssert(result == hipSuccess);
    return result;
}
__host__ hipError_t myCudaFree(void* devPtr, size_t size)
{
    nvAssert(g_nCudaBytes >= size);
    g_nCudaBytes -= size;
    auto result = hipFree(devPtr);
    nvAssert(result == hipSuccess);
    return result;
}

// when we bind buffer for device access, we have to make sure GPU memory is all up-to-date
void GPUBuffer::notifyDeviceBind(bool isWriteBind, bool bDiscardPrevContent)
{
    if (m_hostRev < m_deviceRev)
        return;
    if (m_hostRev > m_deviceRev)
    {
        nvAssert(m_elemSize > 0);
        if (m_nDeviceElems != m_nHostElems)
        {
            if (m_pDevice)
            {
                myCudaFree(m_pDevice, m_nDeviceElems * m_elemSize);
            }
            if (m_nHostElems == 0)
            {
                m_pDevice = nullptr;
            }
            else
            {
                myCudaMalloc((void **)&m_pDevice, m_nHostElems * m_elemSize);
            }
            m_nDeviceElems = m_nHostElems;
        }
        if (!bDiscardPrevContent)
        {
            hipMemcpy(m_pDevice, m_pHost, m_nHostElems * m_elemSize, hipMemcpyHostToDevice);
        }
    }
    m_deviceRev = m_hostRev + (isWriteBind ? 1 : 0);
}
GPUBuffer::~GPUBuffer()
{
    if (m_pDevice)
    {
        nvAssert(m_elemSize > 0);
        myCudaFree(m_pDevice, m_nDeviceElems * m_elemSize);
    }
    delete[](char*)m_pHost;
    m_pHost = nullptr;
}

void GPUBuffer::syncToHost()
{
    if (m_hostRev >= m_deviceRev)
        return;
    nvAssert(m_nHostElems == m_nDeviceElems);
    hipError_t error = hipMemcpy(m_pHost, m_pDevice, m_nHostElems * m_elemSize, hipMemcpyDeviceToHost);
    nvAssert(error == hipSuccess);
    m_hostRev = m_deviceRev;
}

template <class T>
__global__ void clearKernel(T* p, NvU32 nElemsToClear)
{
    NvU32 uElemToClear = blockIdx.x * blockDim.x + threadIdx.x;
    if (uElemToClear >= nElemsToClear)
        return;
    p[uElemToClear] = 0;
}

static inline bool doesRunOnGPU(EXECUTE_MODE mode)
{
    switch (mode)
    {
    case EXECUTE_MODE_DEFAULT:
        return g_bExecuteOnTheGPU;
    case EXECUTE_MODE_FORCE_GPU:
        return true;
    case EXECUTE_MODE_FORCE_CPU:
        return false;
    default:
        nvAssert(false);
        return true;
    }
}

void GPUBuffer::clearSubregion(NvU32 offset, NvU32 nElemsToClear, EXECUTE_MODE mode)
{
    if (doesRunOnGPU(mode))
    {
        notifyDeviceBind(true, nElemsToClear == m_nHostElems);
        dim3 block(256, 1, 1);
        dim3 grid((nElemsToClear + block.x - 1) / block.x, 1, 1);
        if (elemSize() == 4)
        {
            clearKernel << <grid, block >> > (getDevicePointer<float>() + offset, nElemsToClear);
        }
        else
        {
            clearKernel << <grid, block >> > (getDevicePointer<double>() + offset, nElemsToClear);
        }
    }
    else
    {
        nvAssert(offset + nElemsToClear <= size());
        if (elemSize() == 4)
        {
            memset(&(as<float>(offset)), 0, nElemsToClear * elemSize());
        }
        else
        {
            memset(&(as<double>(offset)), 0, nElemsToClear * elemSize());
        }
    }
}

template <class DstType, class SrcType>
__global__ void copyKernel(DstType* pDst, SrcType *pSrc, NvU32 nElems)
{
    NvU32 uElem = blockIdx.x * blockDim.x + threadIdx.x;
    if (uElem >= nElems)
        return;
    pDst[uElem] = (DstType)pSrc[uElem];
}

template <class DstType, class SrcType>
void cpuCopy(DstType* pDst, SrcType* pSrc, NvU32 nElems)
{
    for (NvU32 u = 0; u < nElems; ++u)
    {
        pDst[u] = (DstType)pSrc[u];
    }
}

void GPUBuffer::copySubregionFrom(NvU32 dstOffset, GPUBuffer& src, NvU32 srcOffset, NvU32 nElemsToCopy)
{
    nvAssert(dstOffset + nElemsToCopy <= this->size());
    nvAssert(srcOffset + nElemsToCopy <= src.size());

    if (g_bExecuteOnTheGPU)
    {
        src.notifyDeviceBind(false);
        // (ElemsToCopy == size()) - means we overwrite everything - so can discard the prev content
        notifyDeviceBind(true, nElemsToCopy == size());
        dim3 block(256, 1, 1);
        dim3 grid((nElemsToCopy + block.x - 1) / block.x, 1, 1);
        if (elemSize() == 4)
        {
            float* pDst = getDevicePointer<float>() + dstOffset;
            if (src.elemSize() == 4)
            {
                copyKernel << <grid, block >> > (pDst, src.getDevicePointer<float>() + srcOffset, nElemsToCopy);
            }
            else
            {
                copyKernel << <grid, block >> > (pDst, src.getDevicePointer<double>() + srcOffset, nElemsToCopy);
            }
        }
        else
        {
            double* pDst = getDevicePointer<double>() + dstOffset;
            if (src.elemSize() == 4)
            {
                copyKernel << <grid, block >> > (pDst, src.getDevicePointer<float>() + srcOffset, nElemsToCopy);
            }
            else
            {
                copyKernel << <grid, block >> > (pDst, src.getDevicePointer<double>() + srcOffset, nElemsToCopy);
            }
        }
    }
    else
    {
        syncToHost();
        src.syncToHost();
        nvAssert(m_hostRev >= m_deviceRev);
        m_hostRev = m_deviceRev + 1;
        if (elemSize() == 4)
        {
            float* pDst = &this->as<float>(dstOffset);
            if (src.elemSize() == 4)
            {
                cpuCopy(pDst, &src.as<float>(srcOffset), nElemsToCopy);
            }
            else
            {
                cpuCopy(pDst, &src.as<double>(srcOffset), nElemsToCopy);
            }
        }
        else
        {
            double* pDst = &this->as<double>(dstOffset);
            if (src.elemSize() == 4)
            {
                cpuCopy(pDst, &src.as<float>(srcOffset), nElemsToCopy);
            }
            else
            {
                cpuCopy(pDst, &src.as<double>(srcOffset), nElemsToCopy);
            }
        }
    }
}

double GPUBuffer::autoReadElem(NvU32 uElem)
{
    char buffer[8];
    if (m_deviceRev > m_hostRev)
    {
        hipError_t result = hipMemcpy(buffer, (char *)m_pDevice + uElem * elemSize(), elemSize(), hipMemcpyDeviceToHost);
        nvAssert(result == hipSuccess);
    }
    else
    {
        memcpy(buffer, (char*)m_pDevice + uElem * elemSize(), elemSize());
    }
    return elemSize() == 4 ? *(float*)buffer : *(double*)buffer;
}

void GPUBuffer::autoWriteElem(NvU32 uElem, double value)
{
    char buffer[8];
    if (elemSize() == 4)
    {
        *(float*)buffer = (float)value;
    }
    else
    {
        *(double*)buffer = value;
    }
    if (m_deviceRev > m_hostRev)
    {
        hipError_t result = hipMemcpy((char *)m_pDevice + uElem * elemSize(), buffer, elemSize(), hipMemcpyHostToDevice);
        nvAssert(result == hipSuccess);
    }
    else
    {
        memcpy((char*)m_pHost + uElem * elemSize(), buffer, elemSize());
        m_hostRev = m_deviceRev + 1;
    }
}
