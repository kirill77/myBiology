#include "hip/hip_runtime.h"
#include "neural/l2Computer.h"

// this class collects data from multiple thread of the same block - so it can only run on the GPU
struct CU_LossComputer
{
    static const NvU32 BLOCK_SIZE = 32;

    CU_LossComputer(Tensor& output, Tensor& wantedOutput,
        Tensor& outLoss, GPUBuffer* m_lossPerBlock) : m_output(output),
        m_wantedOutput(wantedOutput), m_outLoss(outLoss, true)
    {
        if (m_lossPerBlock)
        {
            m_errorStat = CUDARWBuffer<float>(*m_lossPerBlock, true);
        }
        nvAssert(wantedOutput.size() == output.size());
        nvAssert(output.size() % output.n() == 0);
        nvAssert(outLoss.size() == output.size());
    }
    __device__ void computeLoss(int threadX, int blockX, int gridDimX)
    {
        int iStride = gridDimX * BLOCK_SIZE;
        float fSumOfSquares = 0;
        int nElements = 0;
        for (int i = blockX * BLOCK_SIZE + threadX; i < m_output.size(); i += iStride)
        {
            float fDiff = m_wantedOutput[i] - m_output[i];
            m_outLoss[i] = fDiff / (m_output.size() / 2.f);
            fSumOfSquares += sqr(fDiff) / m_output.size();
            ++nElements;
        }
        if (m_errorStat.size() == 0)
            return;
        // collect errors from all threads of the block
        const NvU32 FULL_MASK = 0xffffffff;
        for (int offset = 16; offset > 0; offset /= 2)
        {
            fSumOfSquares += __shfl_down_sync(FULL_MASK, fSumOfSquares, offset);
            nElements += __shfl_down_sync(FULL_MASK, nElements, offset);
        }
        if (threadX == 0)
        {
            m_errorStat[blockIdx.x * 2] = fSumOfSquares;
            m_errorStat[blockIdx.x * 2 + 1] = nElements;
        }
    }

private:
    CUDARWTensor<float> m_outLoss;
    CUDAROTensor<float> m_output, m_wantedOutput;
    CUDARWBuffer<float> m_errorStat;
};

__global__ void lossKernel(CU_LossComputer lossComputer)
{
    lossComputer.computeLoss(threadIdx.x, blockIdx.x, gridDim.x);
}

void LossComputer::compute(Tensor& output, Tensor& wantedOutput, Tensor& outLoss, double* pErrorStat)
{
    nvAssert(output.getDims() == wantedOutput.getDims());
    nvAssert(output.getDims() == outLoss.getDims());
    dim3 grid((output.size() + CU_LossComputer::BLOCK_SIZE - 1) / CU_LossComputer::BLOCK_SIZE, 1, 1);
    grid.x = std::min(grid.x, m_lossPerBlock.size() / 2);
    dim3 block(CU_LossComputer::BLOCK_SIZE, 1, 1);
    CU_LossComputer c(output, wantedOutput, outLoss, (pErrorStat == nullptr) ? nullptr : &m_lossPerBlock);
    if (g_bExecuteOnTheGPU)
    {
        lossKernel << <grid, block >> > (c);
        if (pErrorStat)
        {
            double fSumOfSquares = 0;
            int nElements = 0;
            m_lossPerBlock.syncToHost();
            for (NvU32 u = 0; u < grid.x; ++u)
            {
                fSumOfSquares += (double)m_lossPerBlock.as<float>(u * 2);
                nElements += (int)m_lossPerBlock.as<float>(u * 2 + 1);
            }
            *pErrorStat = (float)fSumOfSquares;
        }
    }
    else
    {
        output.syncToHost();
        wantedOutput.syncToHost();
        double fSumOfSquares = 0;
        NvU32 nElements = 0;
        for (NvU32 i = 0; i < outLoss.size(); ++i)
        {
            float fOutput = output.as<float>(i);
            float fDiff = wantedOutput.as<float>(i) - fOutput;
            outLoss.as<float>(i) = fDiff / (output.size() / 2.f);
            fSumOfSquares += sqr(fDiff) / output.size();
            ++nElements;
        }
        if (pErrorStat)
        {
            *pErrorStat = (float)fSumOfSquares;
        }
    }
}

